#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>

using namespace std;

//*******************************************

// Write down the kernels here

__global__ void findNearest(int *xcoord, int *ycoord, int *hp, int *score, int round, int T, int M, int N, int *remainingTanks, int *nearest_arr) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    __shared__ int dist2;
    __shared__ int dirTarget;
    __shared__ int shooterX;
    __shared__ int shooterY;
    __shared__ int targetX;
    __shared__ int targetY;
    __shared__ int quadrant1;

    if(j == 0) {
        dist2 = INT_MAX;
    }
    __syncthreads();
    
    if (j == 0 && hp[i] > 0) {
        dirTarget = (i + round) % T;
        shooterX = xcoord[i];
        shooterY = ycoord[i];
        targetX = xcoord[dirTarget];
        targetY = ycoord[dirTarget];
        if (targetX >= shooterX && targetY >= shooterY)
            quadrant1 = 1;
        else if (targetX <= shooterX && targetY >= shooterY)
            quadrant1 = 2;
        else if (targetX <= shooterX && targetY <= shooterY)
            quadrant1 = 3;
        else
            quadrant1 = 4;
    }
    __syncthreads();

    if(hp[i] > 0) {
        int quadrant2;
        if (xcoord[j] >= shooterX && ycoord[j] >= shooterY)
            quadrant2 = 1;
        else if (xcoord[j] <= shooterX && ycoord[j] >= shooterY)
            quadrant2 = 2;
        else if (xcoord[j] <= shooterX && ycoord[j] <= shooterY)
            quadrant2 = 3;
        else
            quadrant2 = 4;

        if((j != i && hp[j] > 0 && quadrant1 == quadrant2)) {
            int dx = xcoord[j] - shooterX;
            int dy = ycoord[j] - shooterY;
            
            if(((targetY - shooterY) * dx) == (dy * (targetX - shooterX))) {
                int dist = abs(dx) + abs(dy);
                atomicMin(&dist2, dist);
            }
        }
    }
    __syncthreads();

    if(hp[i] > 0) {
        int quadrant2;
        if (xcoord[j] >= shooterX && ycoord[j] >= shooterY)
            quadrant2 = 1;
        else if (xcoord[j] <= shooterX && ycoord[j] >= shooterY)
            quadrant2 = 2;
        else if (xcoord[j] <= shooterX && ycoord[j] <= shooterY)
            quadrant2 = 3;
        else
            quadrant2 = 4;

        if((j != i && hp[j] > 0 && quadrant1 == quadrant2)) {
            int dx = xcoord[j] - shooterX;
            int dy = ycoord[j] - shooterY;
            
            if(((targetY - shooterY) * dx) == (dy * (targetX - shooterX))) {
                int dist = abs(dx) + abs(dy);
                if(dist2 == dist) {
                    nearest_arr[i] = j;
                }
            }
        }
    }
}

__global__ void calculateScore(int *nearest, int *hp, int *remainingTanks, int *score) {
    int i = threadIdx.x;
    if(hp[i] <= 0) return;
    __syncthreads();
    if (nearest[i] != -1) {
        if(atomicAdd(&hp[nearest[i]], -1) == 1) {
            atomicAdd(remainingTanks, -1);
        }
        score[i]++;
    }
}

//***********************************************

int main(int argc, char **argv) {
    // Variable declarations
    int M, N, T, H, *xcoord, *ycoord, *score;

    FILE *inputfilepointer;

    // File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer = fopen(inputfilename, "r");

    if (inputfilepointer == NULL) {
        printf("input.txt file failed to open.");
        return 0;
    }

    fscanf(inputfilepointer, "%d", &M);
    fscanf(inputfilepointer, "%d", &N);
    fscanf(inputfilepointer, "%d", &T); // T is number of Tanks
    fscanf(inputfilepointer, "%d", &H); // H is the starting Health point of each Tank

    // Allocate memory on CPU
    xcoord = (int *)malloc(T * sizeof(int)); // X coordinate of each tank
    ycoord = (int *)malloc(T * sizeof(int)); // Y coordinate of each tank
    score = (int *)malloc(T * sizeof(int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for (int i = 0; i < T; i++) {
        fscanf(inputfilepointer, "%d", &xcoord[i]);
        fscanf(inputfilepointer, "%d", &ycoord[i]);
    }

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int *h_hp;
    h_hp = (int *)malloc(T * sizeof(int));
    for (int i = 0; i < T; i++) {
        h_hp[i] = H;
    }

    int *g_xcoord;
    int *g_ycoord;
    int *g_hp;

    hipMalloc(&g_xcoord, T * sizeof(int));
    hipMalloc(&g_ycoord, T * sizeof(int));
    hipMalloc(&g_hp, T * sizeof(int));

    hipMemcpy(g_xcoord, xcoord, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(g_ycoord, ycoord, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(g_hp, h_hp, T * sizeof(int), hipMemcpyHostToDevice);

    int *g_score;
    hipMalloc(&g_score, T * sizeof(int));

    // Initialize score array to 0
    hipMemset(g_score, 0, T * sizeof(int));

    int round = 1;
    int *remainingTanks = (int *)malloc(sizeof(int));
    int *g_remainingTanks;
    int *g_nearest_arr;

    *remainingTanks = T;

    hipMalloc(&g_remainingTanks, sizeof(int));
    hipMalloc(&g_nearest_arr, T * sizeof(int));
    hipMemcpy(g_remainingTanks, remainingTanks, sizeof(int), hipMemcpyHostToDevice);
    hipMemset(g_nearest_arr, -1, T * sizeof(int));

    while (*remainingTanks > 1) {
        if (round % T != 0) {
            findNearest<<<T, T>>>(g_xcoord, g_ycoord, g_hp, g_score, round, T, M, N, g_remainingTanks, g_nearest_arr);
            calculateScore<<<1, T>>>(g_nearest_arr, g_hp, g_remainingTanks, g_score);
        }
        hipMemcpy(remainingTanks, g_remainingTanks, sizeof(int), hipMemcpyDeviceToHost);
        hipMemset(g_nearest_arr, -1, T * sizeof(int));
        round++;
    }

    hipMemcpy(score, g_score, T * sizeof(int), hipMemcpyDeviceToHost);

    // Free the GPU memory
    hipFree(g_xcoord);
    hipFree(g_ycoord);
    hipFree(g_hp);
    hipFree(g_score);
    hipFree(g_remainingTanks);
    hipFree(g_nearest_arr);
  
    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end - start;

    printf("Execution time : %f\n", timeTaken.count());

    
    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3];
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename, "w");

    for (int i = 0; i < T; i++) {
        fprintf(outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename, "w");
    fprintf(outputfilepointer, "%f", timeTaken.count());
    fclose(outputfilepointer);
    
    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}